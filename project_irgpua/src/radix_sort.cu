#include "hip/hip_runtime.h"
#include "radix_sort.cuh"

#define NB_BINS 10


__global__ void radix_sort_kernel(raft::device_span<int> in, raft::device_span<int> out, int size, int iteration)
{
    int tid = threadIdx.x;
    // Get Blocks Local Histogram
    __shared__ int shared_hist[NB_BINS * 2 + 1]; // assuming NB_BINS elements
    int *inc_cum_hist = &shared_hist[NB_BINS + 1]; // inclusive cum hist
    int *exc_cum_hist = &shared_hist[NB_BINS]; // exclusive cum hist

    if (tid < NB_BINS * 2 + 1)
        shared_hist[tid] = 0;
    __syncthreads();

    if (tid < size)
    {
        int value = in[tid];
        for (int i = 0; i < iteration; ++i)
        {
            value /= NB_BINS;
        }
        atomicAdd(&shared_hist[value % NB_BINS], 1);
    }
    __syncthreads();


    // Load local histogram into inclusive cumulative histogram
    if (tid < NB_BINS)
    {
        inc_cum_hist[tid] = shared_hist[tid];
    }

    __syncthreads();


    // Compute local Exclusive Cumulative Histogram
    for (int i = 1; i < NB_BINS; i *= 2)
    {
        int j = i - 1 + 2 * (tid - tid % i);
        int k = tid % i;
        if (j + k + 1 < NB_BINS)
            inc_cum_hist[j + k + 1] += inc_cum_hist[j];
        __syncthreads();
    }
    __syncthreads();

    if (tid == 0)
        exc_cum_hist[0] = 0;
    
    __syncthreads();
    // Place the values in the correct position
    int my_threads_counter = 0;
    int my_threads_val = tid;
    for (int i = 0; i < size; ++i)
    {
        int base_value = in[i];
        int value = base_value;
        for (int it = 0; it < iteration; ++it)
        {
            value /= NB_BINS;
        }
        if (i < size && (value % NB_BINS) == my_threads_val)
        {
            out[exc_cum_hist[my_threads_val] + my_threads_counter] = base_value;
            my_threads_counter++;
        }
    }

    // And voila ? one round of radix sort done ?
    return;

        __syncthreads();
    if (tid ==0)
    {
        for (int i = 0; i < NB_BINS; ++i)
        {
            out[i] = exc_cum_hist[i];
        }
    }
    __syncthreads();
    return;



}

void radix_sort(rmm::device_uvector<int>& in, rmm::device_uvector<int>& out, int max_value, hipStream_t stream)
{
    int size = in.size();

    //size_t max_threads = 1024;
    //size_t thread_per_block = max_threads;
    //size_t threads = ;
    //size_t blocks = (threads + thread_per_block - 1) / thread_per_block;
    // each thread = 4 pixels and 4 predicate values
    //size_t shared_memory_size = thread_per_block * 4 * 2 * sizeof(int);
    for (int iteration = 0; max_value > 0; iteration++)
    {
        radix_sort_kernel<<<1, size, 0, stream>>>
            (raft::device_span<int>(in.data(), size),
            raft::device_span<int>(out.data(), size),
            size, 
            iteration);
        
            max_value /= NB_BINS;
        hipStreamSynchronize(stream);
        std::swap(in, out);
    }

    std::swap(in, out);
}

// Just run this function to test perfs and correctness
bool test_radix_sort()
{
    int n = 1024; // nb of values in array
    std::vector<int> array(n);
    std::mt19937 rng(42);
    int max = 10000000;
    for (int& x : array)
        x = rng() % max;

    std::vector<int> cpu_array = array;
    std::vector<int> gpu_array = array;
    std::vector<int> cub_array = array;

    // ---------------------------
    // CPU sort timing
    // ---------------------------
    auto start_cpu = std::chrono::high_resolution_clock::now();
    std::sort(cpu_array.begin(), cpu_array.end());
    auto end_cpu = std::chrono::high_resolution_clock::now();
    double cpu_time = std::chrono::duration<double, std::milli>(end_cpu - start_cpu).count();

    // ---------------------------
    // My GPU radix sort
    // ---------------------------
    hipStream_t my_gpu_stream;
    hipStreamCreate(&my_gpu_stream);

    rmm::device_uvector<int> my_gpu_d_in(cpu_array.size(), my_gpu_stream);
    rmm::device_uvector<int> my_gpu_d_out(cpu_array.size(), my_gpu_stream);
    hipMemcpyAsync(my_gpu_d_in.data(), cpu_array.data(), cpu_array.size() * sizeof(int),
                    hipMemcpyHostToDevice, my_gpu_stream);
    auto start_gpu = std::chrono::high_resolution_clock::now();
    radix_sort(my_gpu_d_in, my_gpu_d_out, max - 1, my_gpu_stream);
    auto end_gpu = std::chrono::high_resolution_clock::now();
    double gpu_time = std::chrono::duration<double, std::milli>(end_gpu - start_gpu).count();
    hipMemcpyAsync(gpu_array.data(), my_gpu_d_out.data(), gpu_array.size() * sizeof(int),
                    hipMemcpyDeviceToHost, my_gpu_stream);
    hipStreamDestroy(my_gpu_stream);
    
    // ---------------------------
    // CUB reference GPU sort
    // ---------------------------
    hipStream_t stream;
    hipStreamCreate(&stream);

    rmm::device_uvector<int> d_in(cub_array.size(), stream);
    rmm::device_uvector<int> d_out(cub_array.size(), stream);
    hipMemcpyAsync(d_in.data(), cub_array.data(), cub_array.size() * sizeof(int),
                    hipMemcpyHostToDevice, stream);

    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    // Query temp storage size
    hipcub::DeviceRadixSort::SortKeys(
        d_temp_storage, temp_storage_bytes,
        d_in.data(), d_out.data(), n, 0, sizeof(int) * 8, stream);

    // Allocate temp storage
    hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream);

    auto start_cub = std::chrono::high_resolution_clock::now();
    hipcub::DeviceRadixSort::SortKeys(
        d_temp_storage, temp_storage_bytes,
        d_in.data(), d_out.data(), n, 0, sizeof(int) * 8, stream);
    hipStreamSynchronize(stream);
    auto end_cub = std::chrono::high_resolution_clock::now();

    double cub_time = std::chrono::duration<double, std::milli>(end_cub - start_cub).count();

    hipMemcpyAsync(cub_array.data(), d_out.data(), cub_array.size() * sizeof(int),
                    hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    hipFreeAsync(d_temp_storage, stream);
    hipStreamDestroy(stream);

    // ---------------------------
    // Validation
    // ---------------------------
    bool ok = std::is_sorted(gpu_array.begin(), gpu_array.end());
    bool matches_cpu = (cpu_array == gpu_array);
    bool matches_cub = (cub_array == gpu_array);

    std::cout << "nb values: " << n << "\n";
    std::cout << "max value: " << max << "\n";
    std::cout << "CPU sort time: " << cpu_time << " ms\n";
    std::cout << "My  sort time: " << gpu_time << " ms\n";
    std::cout << "CUB sort time: " << cub_time << " ms\n";

    if (ok && matches_cpu && matches_cub)
        std::cout << "✅ GPU radix sort matches both CPU & CUB results!" << std::endl;
    else
    {
        if (false)
        {
            std::cout << "❌ GPU sort mismatch!\n";
            for (int i = 0; i < n; ++i)
            {
                if (cpu_array[i] != gpu_array[i] || cub_array[i] != gpu_array[i])
                {
                    std::cout << "Index " << i
                            << ": CPU=" << cpu_array[i]
                            << " GPU=" << gpu_array[i]
                            << " CUB=" << cub_array[i] << "\n";
                }
            }
        }
    }

    return ok && matches_cpu && matches_cub;
}