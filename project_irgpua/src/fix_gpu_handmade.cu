#include "hip/hip_runtime.h"
#include "fix_gpu_handmade.cuh"
#include "image.hh"

#include <assert.h>
#include <raft/core/device_span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/device_scalar.hpp>

// Kernel should be launched with 4 values per thread so the mask can be applied without a modulo operation
__global__ void fix_image(raft::device_span<int> buffer) {
    extern __shared__ int shared[];
    int* predicate = &shared[blockDim.x * 4];
    int idx = blockIdx.x * (blockDim.x * 4) + threadIdx.x * 4;
    int size = buffer.size();

    // Load data into shared memory (should coalesce it later)
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        shared[threadIdx.x * 4 + i] = buffer[idx + i];
    }
    __syncthreads();

    // Get Predicate
    constexpr int garbage_val = -27;

    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        if (shared[threadIdx.x * 4 + i] != garbage_val) {
            predicate[threadIdx.x * 4 + i] = 1;
        } else {
            predicate[threadIdx.x * 4 + i] = 0;
        }
    }
    __syncthreads();
    
    // bad scan for tests
    if (threadIdx.x == 0) {
        for (int i = 1; i < blockDim.x * 4; ++i) {
            predicate[i] += predicate[i - 1];
        }

    }
    __syncthreads();

    // NEED TO PROPAGATE SCAN FOR EVERY BLOCK

    // Remove garbage values
    int v0, v1, v2, v3;
    v0 = shared[threadIdx.x * 4 + 0];
    v1 = shared[threadIdx.x * 4 + 1];
    v2 = shared[threadIdx.x * 4 + 2];
    v3 = shared[threadIdx.x * 4 + 3];
    __syncthreads();

    if (v0 != garbage_val) {
        int write_idx = predicate[threadIdx.x * 4 + 0] - 1;
        shared[write_idx] = v0;
    }
    if (v1 != garbage_val) {
        int write_idx = predicate[threadIdx.x * 4 + 1] - 1;
        shared[write_idx] = v1;
    }
    if (v2 != garbage_val) {
        int write_idx = predicate[threadIdx.x * 4 + 2] - 1;
        shared[write_idx] = v2;
    }
    if (v3 != garbage_val) {
        int write_idx = predicate[threadIdx.x * 4 + 3] - 1;
        shared[write_idx] = v3;
    }
    __syncthreads();


}

void fix_image_gpu_handmade(Image& to_fix) { //rmm::device_uvector<int>& buffer, int size) {
    // Kernel is not finished :( But i did radix sort instead so it is worth it !!!
    std::cout << "GPU Handmade version has not been implemented yet !!!! Abort !!" << std::endl;
    return;
}
